#include "hip/hip_runtime.h"
//
// Created by andwh on 12/12/2024.
//

#include "matmul_helpers.cuh"

#include "cpu_matmul.h"

// helper to index into the single ptr array for the 2D matrix
// we are not using pointer-to-pointer approach as it is slower: https://stackoverflow.com/a/53978538
__device__ __host__ inline int idx_in_flattened(int row, int col, int width) {
    return row * width + col;
}

void alloc_matrices(
    float **mat_A, float **mat_B, float **mat_C,
    int m, int n, int k, float
    **d_mat_A, float **d_mat_B, float **d_mat_C) {

    // allocate for host matrices on the cpu
    *mat_A = new float[m * k];
    *mat_B = new float[k * n];
    *mat_C = new float[m * n];

    // allocate device
    hipMalloc(d_mat_A, sizeof(float) * m * k);
    hipMalloc(d_mat_B, sizeof(float) * k * n);
    hipMalloc(d_mat_C, sizeof(float) * n * m);

}

bool verify_against_cpu_matmul(
    const float *h_mat_A, const float *h_mat_B, const float *h_mat_C_gpu,
    const float *h_mat_C_cpu, int M, int N, int K
) {
    // we allocate memory for CPU result
    float *cpu_result = new float[M * N];

    // we run the cpu version
    cpu_matmul(h_mat_A, h_mat_B, cpu_result, M, N, K);

    // lastly verify the result
    for (int i = 0; i < M * N; i++) {
        if (fabs(h_mat_C_gpu[i] - cpu_result[i]) > 1e-5) {
            delete[] cpu_result;
            return false; // cell mismatch - return false
        }
    }

    delete[] cpu_result;
    return true;
}



