#include <iostream>
#include "./matmul_helpers.cuh"
#include "naive_matmul.cuh"

void test_3x2_matmul() {
    int M = 3; // rows of A
    int N = 2; // cols of B
    int K = 2; // cols of A and rows of B

    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    alloc_matrices(&A, &B, &C, &d_A, &d_B, &d_C, M, N, K);

    float A_data[] = {
        1.0f, 2.0f,
        4.0f, 5.0f,
        6.0f, 3.0f
    };

    float B_data[] = {
        1.0f, 2.0f,
        3.0f, 4.0f
    };

    memcpy(A, A_data, M * K * sizeof(float));
    memcpy(B, B_data, K * N * sizeof(float));

    std::cout << "matrix A (3x2):\n";
    print_matrix(A, M, K);

    std::cout << "\nmatrix B (2x2):\n";
    print_matrix(B, K, N);

    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

    naive_matmul(d_A, d_B, d_C, M, N, K);

    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "\nResult Matrix C (3x2):\n";
    print_matrix(C, M, N);

    int is_correct = verify_against_cpu_matmul(A, B, C, M, N, K);
    if (is_correct) {
        std::cout << "Correct matmul kernel" << std::endl;
    } else {
        std::cout << "Incorrect matmul kernel" << std::endl;
    }
}

int main() {
    test_3x2_matmul();
    return 0;
}
